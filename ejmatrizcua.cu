#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void summ(float A[], float B[], float C[], int N){
	int i = threadIdx.x;
//	C[i] = A[i] + B [i];
	printf("%d ", i);
	printf("%.2f \n", A[i]);
}

int main() {
	int N = 3;
	float A[N];
	float B[N];
	float C[N];

	// Inicializar la matriz A
	for (int i = 0; i < N; i++) {
		A[i] = i; // Asignar valores a cada elemento
    	}
	
	// Inicializar la matriz B
	for (int i = 0; i < N; i++) {
		B[i] = i; // Asignar valores a cada elemento
    	}


    	// Imprimir la matriz A
    	for (int i = 0; i < N; i++) {
		printf("%.2f ", A[i]);
    	}
	printf("\n");

	summ<<<2,N>>>(A, B, C, N);
	hipDeviceSynchronize();

    	// Imprimir la matriz C
    	for (int i = 0; i < N; i++) {
		printf("%.2f ", C[i]);
    	}
	printf("\n");


    	return 0;
}

