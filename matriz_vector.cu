#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void vector_producto(float *A, float *v1, float *v2, int matriz_m){
	int fil = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if(col == 0 && fil < matriz_m){
		float sum = 0.0f;
		for(int i = 0; i < matriz_m; i++){
			sum += A[fil * matriz_m + i] * v1[i];
		}
		v2[fil] = sum;
	}
}

int main(int argc, char **argv){
	float *A, *A_gpu;
	float *v1, *v1_gpu;
	float *v2, *v2_gpu;

	int matriz_m = 40000;

	dim3 block_shape = dim3(32,32);
	dim3 grid_shape = dim3(max(1.0, ceil((float)matriz_m / (float) block_shape.x)),
			max(1.0, ceil((float)matriz_m / (float) block_shape.x)));
	
	A = (float *)malloc(matriz_m * matriz_m * sizeof(float));
	v1 = (float *)malloc(matriz_m * sizeof(float));
	v2 = (float *)malloc(matriz_m * sizeof(float));

	for(int i = 0; i < matriz_m; i++){
		for(int j = 0; j < matriz_m; j++){
			A[i * matriz_m + j] = (float) i * matriz_m + j;
			//printf("A[i] = %.2f\n", A[i * matriz_m + j]);
		}
	}

	for(int i = 0; i < matriz_m; i++){
		v1[i] = (float) i;
		//printf("v1[i] = %.2f\n", v1[i]);
	}

	hipMalloc((void **) &A_gpu, matriz_m * matriz_m * sizeof(float));
	hipMalloc((void **) &v1_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &v2_gpu, matriz_m * sizeof(float));

	hipMemcpy(A_gpu, A, matriz_m * matriz_m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(v1_gpu, v1, matriz_m * sizeof(float), hipMemcpyHostToDevice);

	vector_producto<<<grid_shape, block_shape>>>(A_gpu, v1_gpu, v2_gpu, matriz_m);

	hipMemcpy(v2, v2_gpu, matriz_m * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < matriz_m; i ++){
		printf("v2 = %.2f\n", v2[i]);
	}

	free(A);
	free(v1);
	free(v2);

	hipFree(A_gpu);
	hipFree(v1_gpu);
	hipFree(v2_gpu);

	return 0;
}
