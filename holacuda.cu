#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void printKernel(){
        printf("Hola CUDA\n");
	printf("ID Bloque X: %d, ID Bloque Y: %d, ID Hilo X: %d, ID Hilo Y: %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}       

int main(){
	// <<<Bloque,Hilo>>>
        printKernel<<<2,2>>>();
	hipDeviceSynchronize();
	return 0;
}  
