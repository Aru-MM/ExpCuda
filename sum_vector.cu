#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_producto(float *A, float *B, float *C, int matriz_m){
	//printf("BIdx BDmx HIdx BIdy BDmy HIdy BIdz BDmz HIdz \n");
	//    D B H      D B H      D B H
	// x[ 1 1 1 ] y[ 1 1 1 ] z[ 1 1 1 ]
	//printf("x[ %d %d %d %d ] y[ %d %d %d %d ] z[ %d %d %d %d ] \n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x, gridDim.y, blockDim.y, blockIdx.y, threadIdx.y, gridDim.z, blockDim.z, blockIdx.z, threadIdx.z);
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//printf("i = %d\n", i);

	if(i < matriz_m){
		C[i] = A[i] + B[i];
		//printf("C = %.2f\n", C[i]);
	}
}

int main(int argc, char **argv){
	
	//Numero de elementos y su tamaño
	int matriz_m = 400000;

	// Variables de locales
	float *A = (float *)malloc(matriz_m * sizeof(float));
	float *B = (float *)malloc(matriz_m * sizeof(float));
	float *C = (float *)malloc(matriz_m * sizeof(float));

	//Inicializa vectores locales
	//printf("A = ");
	for(int i = 0; i < matriz_m; i++){
		A[i] = (float) i;
		//printf("%.2f ", A[i]);
	}
	//printf("\n");

	//printf("B = ");
	for(int i = 0; i < matriz_m; i++){
		B[i] = (float) i;
		//printf("%.2f ", B[i]);
	}
	//printf("\n");

	//Variables GPU
	float *A_gpu;
	float *B_gpu;
	float *C_gpu;

	hipMalloc((void **) &A_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &B_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &C_gpu, matriz_m * sizeof(float));

	//Copia las variables locales a la gpu
	hipMemcpy(A_gpu, A, matriz_m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, matriz_m * sizeof(float), hipMemcpyHostToDevice);

	//Lanzar el kernel de suma
	//dim3 block_shape = dim3(32,32);
	//dim3 grid_shape = dim3(max(1.0, ceil((float)matriz_m / (float) block_shape.x)),
			//max(1.0, ceil((float)matriz_m / (float) block_shape.x)));
	
	dim3 BpR = dim3(65535,1,1);
	dim3 HpB = dim3(1024,1,1);

	//int hilosporbloque = 10;
	//int bloquespormalla = (matriz_m + hilosporbloque - 1) / hilosporbloque;
	
	//printf("BS = [ %d, %d, %d ] GS = [ %d, %d, %d ]\n", block_shape.x, block_shape.y, block_shape.z, grid_shape.x, grid_shape.y, grid_shape.z);
	//printf("HB = %d BC = %d\n", hilosporbloque, bloquespormalla);
	//printf("BpR = [ %d, %d, %d ] HpB = [ %d, %d, %d ]\n", BpR.x, BpR.y, BpR.z, HpB.x, HpB.y, HpB.z);

	//printf("   D B H      D B H      D B H\n");
	//vector_producto<<<grid_shape, block_shape>>>(A_gpu, B_gpu, C_gpu, matriz_m);
	//vector_producto<<<bloquespormalla, hilosporbloque>>>(A_gpu, B_gpu, C_gpu, matriz_m);
	vector_producto<<<BpR, HpB>>>(A_gpu, B_gpu, C_gpu, matriz_m);
	
	hipDeviceSynchronize();
	
	//Copia las variables de la GPU a local
	hipMemcpy(C, C_gpu, matriz_m * sizeof(float), hipMemcpyDeviceToHost);

	//printf("C = ");
	//for(int i = 0; i < matriz_m; i ++){
		//printf("%.2f ", C[i]);
	//}
	//printf("\n");

	//Libera la memoria local
	free(A);
	free(B);
	free(C);

	//Libera la memoria GPU
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);

	return 0;
}
