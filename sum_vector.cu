#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void vector_producto(float *A, float *B, float *C, int matriz_m){
	printf("BId = %d BDm = %d HId = %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	int i = threadIdx.x;

	C[i] = A[i] + B[i];
}

int main(int argc, char **argv){
	float *A, *A_gpu;
	float *B, *B_gpu;
	float *C, *C_gpu;

	int matriz_m = 4;

	dim3 block_shape = dim3(3,3);
	dim3 grid_shape = dim3(max(1.0, ceil((float)matriz_m / (float) block_shape.x)),
			max(1.0, ceil((float)matriz_m / (float) block_shape.x)));
	
	A = (float *)malloc(matriz_m * sizeof(float));
	B = (float *)malloc(matriz_m * sizeof(float));
	C = (float *)malloc(matriz_m * sizeof(float));

	printf("A = ");
	for(int i = 0; i < matriz_m; i++){
		A[i] = (float) i;
		printf("%.2f ", A[i]);
	}
	printf("\n");

	printf("B = ");
	for(int i = 0; i < matriz_m; i++){
		B[i] = (float) i;
		printf("%.2f ", B[i]);
	}
	printf("\n");

	hipMalloc((void **) &A_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &B_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &C_gpu, matriz_m * sizeof(float));

	hipMemcpy(A_gpu, A, matriz_m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, matriz_m * sizeof(float), hipMemcpyHostToDevice);

	vector_producto<<<2, 4>>>(A_gpu, B_gpu, C_gpu, matriz_m);

	hipMemcpy(C, C_gpu, matriz_m * sizeof(float), hipMemcpyDeviceToHost);

	printf("C = ");
	for(int i = 0; i < matriz_m; i ++){
		printf("%.2f ", C[i]);
	}
	printf("\n");

	free(A);
	free(B);
	free(C);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);

	return 0;
}
