#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_producto(float *A, float *B, float *C, int matriz_m){
//	printf("BId = %d BDm = %d HId = %d \n", blockIdx.x, blockDim.x, threadIdx.x);
	printf("BId = %d BDm = %d HId = %d \n", blockIdx.y, blockDim.y, threadIdx.y);
	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	printf("i = %d\n", i);

	if(i < matriz_m){
		C[i] = A[i] + B[i];
		printf("C = %.2f\n", C[i]);
	}
}

int main(int argc, char **argv){
	
	//Numero de elementos y su tamaño
	int matriz_m = 4;

	// Variables de locales
	float *A = (float *)malloc(matriz_m * sizeof(float));
	float *B = (float *)malloc(matriz_m * sizeof(float));
	float *C = (float *)malloc(matriz_m * sizeof(float));

	//Inicializa vectores locales
	printf("A = ");
	for(int i = 0; i < matriz_m; i++){
		A[i] = (float) i;
		printf("%.2f ", A[i]);
	}
	printf("\n");

	printf("B = ");
	for(int i = 0; i < matriz_m; i++){
		B[i] = (float) i;
		printf("%.2f ", B[i]);
	}
	printf("\n");

	//Variables GPU
	float *A_gpu;
	float *B_gpu;
	float *C_gpu;

	hipMalloc((void **) &A_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &B_gpu, matriz_m * sizeof(float));
	hipMalloc((void **) &C_gpu, matriz_m * sizeof(float));

	//Copia las variables locales a la gpu
	hipMemcpy(A_gpu, A, matriz_m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, matriz_m * sizeof(float), hipMemcpyHostToDevice);

	//Lanzar el kernel de suma
	dim3 block_shape = dim3(32,32);
	dim3 grid_shape = dim3(max(1.0, ceil((float)matriz_m / (float) block_shape.x)),
			max(1.0, ceil((float)matriz_m / (float) block_shape.x)));
	
	int hilosporbloque = 4;
	int bloquespormalla = (matriz_m + hilosporbloque - 1) / hilosporbloque;
	
	printf("BS = %d, %d, %d GS = %d, %d, %d\n", block_shape.x, block_shape.y, block_shape.z, grid_shape.x, grid_shape.y, grid_shape.z);
	printf("HB = %d BC = %d\n", hilosporbloque, bloquespormalla);

	hipMemcpy(A_gpu, A, matriz_m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, matriz_m * sizeof(float), hipMemcpyHostToDevice);
	//vector_producto<<<grid_shape, block_shape>>>(A_gpu, B_gpu, C_gpu, matriz_m);
	vector_producto<<<bloquespormalla, hilosporbloque>>>(A_gpu, B_gpu, C_gpu, matriz_m);

	//Copia las variables de la GPU a local
	hipMemcpy(C, C_gpu, matriz_m * sizeof(float), hipMemcpyDeviceToHost);

	printf("C = ");
	for(int i = 0; i < matriz_m; i ++){
		printf("%.2f ", C[i]);
	}
	printf("\n");

	//Libera la memoria local
	free(A);
	free(B);
	free(C);

	//Libera la memoria GPU
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);

	return 0;
}
